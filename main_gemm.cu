#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FETCH_FLOAT4(float_var) (reinterpret_cast<float4*>(&(float_var))[0])
#define CUDA_CHECK(call)                                                       \
    do {                                                                       \
        hipError_t err = call;                                                \
        if (err != hipSuccess) {                                              \
            printf("CUDA Error: \n");                                          \
            printf("    File:       %s\n", __FILE__);                          \
            printf("    Line:       %d\n", __LINE__);                          \
            printf("    Error Code: %d\n", err);                               \
            printf("    Error Text: %s\n", hipGetErrorString(err));           \
            exit(1);                                                           \
        }                                                                      \
    } while (0)

void gemm_cpu(float* A, float* B, float* C, int M, int N, int K){
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            for (int k = 0; k < K; k++){
                C[i * N + j] += A[i * K + k] * B[k * N + j]; 
            }
        }
    }
}

void init_gemm_random(float* A, int M, int N){
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            A[i * N + j] = static_cast<float>(rand()) / RAND_MAX;
        }
    }
}

void __global__ gemm_kernel(float *A, float *B, float *C, int M, int N, int K){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < M && col < N){
        float sum = 0.0f;
        for (int k = 0; k < K; k++){
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void launch_kernel_1(){
    int M = 1024;
    int N = 1024;
    int K = 1024;
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
    float* h_A = (float*)malloc(size_A);
    float* h_B = (float*)malloc(size_B);
    float* h_C = (float*)malloc(size_C);

    init_gemm_random(h_A, M, K);
    init_gemm_random(h_B, K, N);

    float *d_A , *d_B, *d_C;
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    int TN = 16;
    int TM = 32;
    dim3 blockSize(TN, TM);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);
    
    //kernel 计时
    int warm_up = 2;
    int repeat = 10;
    for (int i = 0; i < warm_up; i++){
        gemm_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
        hipDeviceSynchronize();
    }

    float kernel_time_sum = 0.0f;
    for (int i = 0; i < repeat; i++){
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        gemm_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float kernel_time = 0.0f;
        hipEventElapsedTime(&kernel_time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        kernel_time_sum += kernel_time;
    }
    printf("CUDA kernel time is : %.3f ms\n", kernel_time_sum / repeat);
    

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
    float *h_C_cpu = (float*)malloc(size_C);

    //CPU计时
    auto cpu_start = std::chrono::high_resolution_clock::now();
    gemm_cpu(h_A, h_B, h_C_cpu, M, N, K);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    double cpu_time = std::chrono::duration<double, std::milli>(cpu_end - cpu_start).count();
    printf("CPU gemm time: %.3f ms\n", cpu_time);
    
    float max_diff = 0.0f;
    for (int i = 0; i < M * N; i++){
        if (std::fabs(h_C[i] - h_C_cpu[i]) > max_diff){
            max_diff = std::fabs(h_C[i] - h_C_cpu[i]);
        }
    }
    printf("max diff is : %f\n", max_diff);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cpu);
}

__global__ void sgemm_gpu_kernel_v2(float *__restrict__ A,
                                    float *__restrict__ B,
                                    float *__restrict__ C, const int M,
                                    const int N, const int K) {
    const int BM = 16, BN = 16;
    const int BK = 64;
    __shared__ float s_a[BM][BK], s_b[BK][BN];
    float c = 0.0f;

    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    // 每次从全局内存加载到共享内存，每个线程都负责一个float4。以下是当前线程负责的这个float4的索引
    const int row_s_a = tid / 16;
    const int col_s_a = (tid % 16) * 4;
    const int row_s_b = tid / 4;
    const int col_s_b = (tid % 4) * 4;
    // 每个线程从读取的全局内存的位置，在A上的行是固定不变的，在B上列是固定不变的
    const int row_A = blockIdx.y * BM + row_s_a;
    const int col_B = blockIdx.x * BN + col_s_b;

    for (int step = 0; step < K / BK; step++) {
        // 从A加载到s_a
        const int col_A = step * BK + col_s_a;
        const int index_A = OFFSET(row_A, col_A, K);
        FETCH_FLOAT4(s_a[row_s_a][col_s_a]) = FETCH_FLOAT4(A[index_A]);
        // 从B加载到s_b
        const int row_B = step * BK + row_s_b;
        const int index_B = OFFSET(row_B, col_B, N);
        FETCH_FLOAT4(s_b[row_s_b][col_s_b]) = FETCH_FLOAT4(B[index_B]);
        __syncthreads();
        // 计算
        for (int k = 0; k < BK; k++) {
            const float a = s_a[threadIdx.y][k];
            const float b = s_b[k][threadIdx.x];
            c += a * b;
        }
        __syncthreads();
    }
    // 写入C
    const int row_C = blockIdx.y * BM + threadIdx.y;
    const int col_C = blockIdx.x * BN + threadIdx.x;
    const int index_C = OFFSET(row_C, col_C, N);
    C[index_C] = c;
}

void __global__ gemm_kernel_fix_2(float* A, float* B, float* C, 
    const int M, const int N, const int K){
    const int BM = 16;
    const int BN = 16;
    const int BK = 64;
    __shared__ float sm_a[BM][BK];
    __shared__ float sm_b[BK][BN];

    int row = blockIdx.y * BM + threadIdx.y;
    int col = blockIdx.x * BN + threadIdx.x;
    float c = 0.0f;

    for (int step = 0; step < K / BK; ++step) {
        // load A block to shared memory
        for (int k = threadIdx.x; k < BK; k += BN) {
            if (row < M && (step * BK + k) < K)
                sm_a[threadIdx.y][k] = A[row * K + step * BK + k];
            else
                sm_a[threadIdx.y][k] = 0.0f;
        }
        // load B block to shared memory
        for (int k = threadIdx.y; k < BK; k += BM) {
            if ((step * BK + k) < K && col < N)
                sm_b[k][threadIdx.x] = B[(step * BK + k) * N + col];
            else
                sm_b[k][threadIdx.x] = 0.0f;
        }
        __syncthreads();

        for (int k = 0; k < BK; ++k) {
            c += sm_a[threadIdx.y][k] * sm_b[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < M && col < N)
        C[row * N + col] = c;

}

void __global__ gemm_kernel_2(float* A, float* B, float* C, 
    const int M, const int N, const int K){
    const int BM = 16;
    const int BN = 16;
    const int BK = 64;
    __shared__ float sm_a[BM][BK];
    __shared__ float sm_b[BK][BN];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int row_s_a = tid / 16;
    const int col_s_a = (tid % 16) * 4;
    const int row_s_b = tid / 4;
    const int col_s_b = (tid % 4) * 4;
    // int row_s_a = threadIdx.y;
    // int col_s_a = threadIdx.x * 4;
    // int row_s_b = threadIdx.y * 4;
    // int col_s_b = (threadIdx.x % 4) * 4;

    int row_A = row_s_a + BM * blockIdx.y;
    int col_B = col_s_b + BN * blockIdx.x;

    int row_C = row_A;
    int col_C = col_B;
    float c = 0.0f;

    for (int step = 0; step < (K / BK); step++){
        int col_A = step * BK + col_s_a;
        int row_B = step * BK + row_s_b;

        FETCH_FLOAT4(sm_a[row_s_a][col_s_a]) = FETCH_FLOAT4(A[row_A * K + col_A]);
        FETCH_FLOAT4(sm_b[row_s_b][col_s_b]) = FETCH_FLOAT4(B[row_B * N + col_B]);
        __syncthreads();
        for (int k = 0; k < BK; k++){
            c += sm_a[row_s_a][k] * sm_b[k][col_s_b];
        }
    }
    C[row_C * N + col_C] = c;

}

void launch_kernel_2(){
    int M = 1024;
    int N = 1024;
    int K = 1024;
    int BM = 16;
    int BN = 16;
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);
    float* h_A = (float*)malloc(size_A);
    float* h_B = (float*)malloc(size_B);
    float* h_C = (float*)malloc(size_C);

    init_gemm_random(h_A, M, K);
    init_gemm_random(h_B, K, N);

    float *d_A , *d_B, *d_C;
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    
    dim3 blockSize(BN, BM);
    dim3 gridSize((N + BN - 1) / BN, (M + BM - 1) / BM);

    int warm_up = 2;
    int repeat = 5;
    for (int i = 0; i < warm_up; i++){
        gemm_kernel_fix_2<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }
    float kernel_time_sum = 0.0f;
    for (int i = 0; i < repeat; i++){
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        gemm_kernel_fix_2<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float kernel_time = 0.0f;
        hipEventElapsedTime(&kernel_time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        kernel_time_sum += kernel_time;
    }
    printf("kernel_2_time_sum is : %.3fms\n", kernel_time_sum / repeat);
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);
    float *h_C_cpu = (float*)malloc(size_C);

    //CPU计时
    auto cpu_start = std::chrono::high_resolution_clock::now();
    gemm_cpu(h_A, h_B, h_C_cpu, M, N, K);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    double cpu_time = std::chrono::duration<double, std::milli>(cpu_end - cpu_start).count();
    printf("CPU gemm time: %.3f ms\n", cpu_time);
    
    float max_diff = 0.0f;
    for (int i = 0; i < M * N; i++){
        if (std::fabs(h_C[i] - h_C_cpu[i]) > max_diff){
            max_diff = std::fabs(h_C[i] - h_C_cpu[i]);
        }
    }
    printf("max diff is : %f\n", max_diff);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cpu);
    
}

int main(){
    launch_kernel_1();
    printf("=============\n");
    launch_kernel_2();
    return 0;
}